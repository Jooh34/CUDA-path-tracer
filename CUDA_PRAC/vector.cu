#include "hip/hip_runtime.h"
#include "vector.cuh"

Vec::Vec(double x, double y, double z) {
    this->x = x;
    this->y = y;
    this->z = z;
}

Vec Vec::operator+(const Vec &b) {
     return Vec(x+b.x,y+b.y,z+b.z);
}

Vec Vec::operator-(const Vec &b) {
     return Vec(x-b.x,y-b.y,z-b.z);
}

Vec Vec::operator*(double b) {
    return Vec(x*b,y*b,z*b);
}

Vec Vec::cross(const Vec &b) {
    return Vec(y*b.z-z*b.y,z*b.x-x*b.z,x*b.y-y*b.x);
}

double Vec::dot(const Vec &b) {
    return x*b.x+y*b.y+z*b.z;
}

Vec& Vec::norm() {
    return *this = *this * (1/sqrt(x*x+y*y+z*z));
}
