#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "ray.cuh"
#include "vector.cuh"
#include "camera.cuh"

#include <vector>
#include <stdio.h>
#include <iostream>

#include <stdio.h>
#include <ctime>
#include <hiprand/hiprand_kernel.h>

#define M_PI 3.141592

#include <iostream>

using namespace std;

__global__ void initializeRays(Camera* cam, Ray* rays) {// initialize rays on the host
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * cam->width);

	hiprandState state;
	hiprand_init((unsigned long long)clock() + index, 0, 0, &state);
	hiprand_uniform_double(&state);

	Vec center = cam->position + cam->direction * cam->vp_dist;
	double vp_width = 2 * cam->vp_dist * tan((cam->FOV / 2) * M_PI / 180.0);
	double vp_height = 1. / cam->ratio * vp_width;

	double px_width = vp_width / cam->width;
	double px_height = vp_height / cam->height;

	Vec start_pixel = center - cam->x_direction * (vp_width / 2) + cam->y_direction * (vp_height / 2);
	Vec target_pixel = start_pixel + cam->x_direction * px_width * (x + 0.5) - cam->y_direction * px_height * (y + 0.5);

	// jitter for anti-aliasing
	double jitter_x = hiprand_uniform_double(&state)-0.5;
	double jitter_y = hiprand_uniform_double(&state)-0.5;

	Vec jittered_target = target_pixel + cam->x_direction * jitter_x * px_width + cam->y_direction * jitter_y * px_height;
	//

	double e1 = hiprand_uniform_double(&state)-0.5;
	double e2 = hiprand_uniform_double(&state)-0.5;
	double e3 = hiprand_uniform_double(&state)-0.5;
	double d = cam->radius * hiprand_uniform_double(&state);

	Vec rand_vec = Vec(e1, e2, e3);

	Vec orth = (cam->direction.cross(rand_vec)).norm();
	Vec jittered_position = cam->position + orth * d;
	//
	rays[index].origin = jittered_position;
	rays[index].direction = (jittered_target - jittered_position).norm();
}

int main() {
    int N = 500;

	Vec origin(0, 500, 1400);
    Vec dest(0, 500, 0);
	Camera* camera = new Camera(origin, dest, 500, 500, 1400, 60);
	Camera* cuda_camera;

	hipMalloc(&cuda_camera, sizeof(Camera));
	hipMemcpy(cuda_camera, camera, sizeof(Camera), hipMemcpyHostToDevice);

    Ray *rays;
	Vec *colors;

	hipMallocManaged(&rays, N * sizeof(Ray));
	hipMallocManaged(&colors, N * sizeof(Vec));

    initializeRays <<<N, 1>>> (cuda_camera, rays);

	hipDeviceSynchronize();

	for (int i = 0; i < 500; i++) {
		cout << rays[i].direction.x << ", " << rays[i].direction.y << ", " << rays[i].direction.z << endl;
	}
}
