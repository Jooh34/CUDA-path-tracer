#include "hip/hip_runtime.h"
#include "vector.cuh"
#include "camera.cuh"
#include "ray.cuh"
#include <iostream>

using namespace std;

Camera::Camera(Vec position, Vec target, int width, int height, float vp_dist, int FOV) {
    this->position = position;
    this->target = target;
    this->width = width;
    this->height = height;
    this->vp_dist = vp_dist;
    this->FOV = FOV;

    ratio = (double)width/(double)height;
    direction = (target - position).norm();

    look_up = Vec(0, 1, 0);
    radius = 0;

    x_direction = direction.cross(look_up).norm();
    y_direction = x_direction.cross(direction).norm();
}
